#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "dataStructs.h"
#include <cfloat>

__device__ int squaredDistance(point_t a, point_t b) {
    return (a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y);
}

__global__ void findNearestNodeKernel(point_t coordinate, node_t *list_of_nodes, int num_of_nodes, int *min_d2, int *min_index) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_of_nodes) {
        int d2 = squaredDistance(coordinate, list_of_nodes[i].point);
        atomicMin(min_d2, d2);
        if (*min_d2 == d2) {
            *min_index = i;
        }
    }
}

bool findNearestNodeToCoordinateCUDA(point_t coordinate, node_t *list_of_nodes, int num_of_nodes, node_t **nearest_node,
                                     int map_dim_x, int map_dim_y, int dist_to_grow) {
    node_t *d_list_of_nodes;
    int *d_min_d2, *d_min_index;
    int min_d2 = pow(map_dim_x + map_dim_y, 2);
    int min_index = -1;

    // Allocate memory on the device
    hipMalloc((void **)&d_list_of_nodes, num_of_nodes * sizeof(node_t));
    hipMalloc((void **)&d_min_d2, sizeof(int));
    hipMalloc((void **)&d_min_index, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_list_of_nodes, list_of_nodes, num_of_nodes * sizeof(node_t), hipMemcpyHostToDevice);
    hipMemcpy(d_min_d2, &min_d2, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_min_index, &min_index, sizeof(int), hipMemcpyHostToDevice);

    // Kernel launch code - assuming 1024 threads per block
    int threadsPerBlock = 1024;
    int blocksPerGrid = (num_of_nodes + threadsPerBlock - 1) / threadsPerBlock;
    findNearestNodeKernel<<<blocksPerGrid, threadsPerBlock>>>(coordinate, d_list_of_nodes, num_of_nodes, d_min_d2, d_min_index);

    // Copy result back to host
    hipMemcpy(&min_d2, d_min_d2, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&min_index, d_min_index, sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_list_of_nodes);
    hipFree(d_min_d2);
    hipFree(d_min_index);

    *nearest_node = &list_of_nodes[min_index];

    return min_d2 >= dist_to_grow;
}